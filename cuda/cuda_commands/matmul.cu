#include "hip/hip_runtime.h"
//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "runtime/ext/cuda/cuda_commands.h"
#include "cuda/cuda_buffer.h"
#include "cuda/math/vec.h"

namespace luisa::compute::cuda {
template<typename TYPE>
__global__ void matmul_tiled_fp32(hipDeviceptr_t src0, hipDeviceptr_t src1, hipDeviceptr_t dst, uint32_t kLoopSize) {
    auto *inputA = reinterpret_cast<TYPE *>(src0);
    auto *inputB = reinterpret_cast<TYPE *>(src1);
    auto *outputO = reinterpret_cast<TYPE *>(dst);

    auto index = grid_index();

    TYPE a = inputA[index];
    TYPE b = inputB[index];
    TYPE c = TYPE(1.f, 1.f, 1.f, 1.f);
    for (int i = 0; i < kLoopSize; i++) {
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
        c = cw_mul(a, c) + b;
    }
    outputO[index] = c;
}

CudaCommand::UCommand CudaCommand::matmul(BufferView<float> src0_buffer, BufferView<float> src1_buffer, BufferView<float> dst_buffer,
                                          int tileM, int tileN, int tileK,
                                          int M, int N, int K,
                                          int wg_size_x, int wg_size_y) noexcept {
    return luisa::make_unique<luisa::compute::cuda::CudaLCubCommand>(
        [=](hipStream_t stream) {
            dim3 gridDim(uint32_t(N / tileN), uint32_t(M / tileM));
            dim3 blockDim(wg_size_x, wg_size_y);
            matmul_tiled_fp32<vec4f><<<gridDim, blockDim, 0, stream>>>(
                reinterpret_cast<const CUDABuffer *>(src0_buffer.handle())->handle(),
                reinterpret_cast<const CUDABuffer *>(src1_buffer.handle())->handle(),
                reinterpret_cast<const CUDABuffer *>(dst_buffer.handle())->handle(),
                src0_buffer.size());
        });
}

}// namespace luisa::compute::cuda