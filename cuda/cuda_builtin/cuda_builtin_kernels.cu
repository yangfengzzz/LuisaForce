//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include <hip/hip_runtime.h>

// built-in update kernel for BindlessArray
struct alignas(16u) BindlessSlot {
    unsigned long long buffer;
    unsigned long long buffer_size;
    unsigned long long tex2d;
    unsigned long long tex3d;
};

static_assert(sizeof(BindlessSlot) == 32u, "");

struct alignas(16) SlotModification {
    struct Buffer {
        unsigned long long handle;
        unsigned long long size;
        unsigned int op;
    };
    struct Texture {
        unsigned long long handle;
        unsigned int sampler;// not used; processed on host
        unsigned int op;
    };
    unsigned long long slot;
    Buffer buffer;
    Texture tex2d;
    Texture tex3d;
};

static_assert(sizeof(SlotModification) == 64u, "");

__global__ void update_bindless_array(BindlessSlot *__restrict__ array,
                                      const SlotModification *__restrict__ mods,
                                      unsigned int n) {
    constexpr auto op_update = 1u;
    constexpr auto op_remove = 2u;
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) [[likely]] {
        auto m = mods[tid];
        auto slot_id = m.slot;
        auto slot = array[slot_id];
        if (m.buffer.op == op_update) {
            slot.buffer = m.buffer.handle;
            slot.buffer_size = m.buffer.size;
        } else if (m.buffer.op == op_remove) {
            slot.buffer = 0u;
            slot.buffer_size = 0u;
        }
        if (m.tex2d.op == op_update) {
            slot.tex2d = m.tex2d.handle;
        } else if (m.tex2d.op == op_remove) {
            slot.tex2d = 0u;
        }
        if (m.tex3d.op == op_update) {
            slot.tex3d = m.tex3d.handle;
        } else if (m.tex3d.op == op_remove) {
            slot.tex3d = 0u;
        }
        array[slot_id] = slot;
    }
}

void update_bindless_array(hipStream_t hip_stream, hipDeviceptr_t array, hipDeviceptr_t mods, uint32_t n) {
    update_bindless_array<<<dim3((n + 255u) / 256u, 1u, 1u),
                            dim3(256u, 1u, 1u), 0, hip_stream>>>(
        reinterpret_cast<BindlessSlot *>(array),
        reinterpret_cast<SlotModification *>(mods), n);
}